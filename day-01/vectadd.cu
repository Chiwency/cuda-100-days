
#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check_cuda_error((val), #val, __FILE__, __LINE__)
void check_cuda_error(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result != hipSuccess) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" 
                  << func << "' -> " << hipGetErrorString(result) << std::endl;
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

using namespace std;

int main()
{
    const int N = 10;
    int *a, *b, *c;

    hipMallocManaged(&a, N * sizeof(int));
    hipMallocManaged(&b, N * sizeof(int));
    hipMallocManaged(&c, N * sizeof(int));
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    int blockSize = 1024;
    int gridSize = (N + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(a, b, c, N);
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR(hipGetLastError());
    for (int i = 0; i < N; i++)
    {
        cout << c[i] << " ";
    }

    cout << endl;
    return 0;
}